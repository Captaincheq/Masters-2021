
#include "Define.h"
#include "nvToolsExt.h"

namespace ckks {

void CudaNvtxStart(std::string msg) { nvtxRangePushA(msg.c_str()); }
void CudaNvtxStop() { nvtxRangePop(); }
void CudaHostSync() { hipDeviceSynchronize(); }

}