
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <rmm/device_buffer.hpp>
#include <rmm/device_uvector.hpp>

#include "DeviceVector.h"

using namespace ckks;

void DeviceVector::append(const DeviceVector& out) {
  size_t old_size = size();
  resize(size() + out.size());
  hipMemcpyAsync(data() + old_size, out.data(), out.size() * sizeof(Dtype),
                  hipMemcpyDefault, stream_);
}